#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "cuda_error_check.cuh"
#include <glm/glm.hpp>
#include "util.h"

__global__ void voxelize_triangle(voxinfo info, float* triangle_data, bool* voxel_table){
	size_t thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	//printf("Thread %i saying hi \n", thread_id);
	
	using namespace glm; // we use GLM for all the vector operations

	while(thread_id < info.n_triangles){ // every thread works on specific triangles
		//printf("Looking at triangle %i \n", thread_id);
		size_t t = thread_id*9; // triangle contains 9 vertices

		// COMPUTE COMMON TRIANGLE PROPERTIES
		vec3 delta_p = vec3(info.unitlength, info.unitlength, info.unitlength);
		vec3 v0 = vec3(triangle_data[t], triangle_data[t+1], triangle_data[t+2]);
		vec3 v1 = vec3(triangle_data[t+3], triangle_data[t+4], triangle_data[t+5]);
		vec3 v2 = vec3(triangle_data[t+6], triangle_data[t+7], triangle_data[t+8]);
		vec3 e0 = v1-v0;
		vec3 e1 = v2-v1;
		vec3 e2 = v0-v2;
		vec3 n = normalize(cross(e0,e1));

		// PREPARE PLANE TEST PROPERTIES
		vec3 c = vec3(0.0f, 0.0f, 0.0f); // critical point
		if (n.x > 0) { c.x = info.unitlength;}
		if (n.y > 0) { c.y = info.unitlength;}
		if (n.z > 0) { c.z = info.unitlength;}
		float d1 = dot(n, (c - v0));
		float d2 = dot(n, ((delta_p - c) - v0));

		// PREPARE PROJECTION TEST PROPERTIES
		// XY plane
		vec2 n_xy_e0 = vec2(-1.0f*e0.y, e0.x);
		vec2 n_xy_e1 = vec2(-1.0f*e1.y, e1.x);
		vec2 n_xy_e2 = vec2(-1.0f*e2.y, e2.x);
		if (n.z < 0.0f) {
			n_xy_e0 = -n_xy_e0;
			n_xy_e1 = -n_xy_e1;
			n_xy_e2 = -n_xy_e2;
		}
		float d_xy_e0 = (-1.0f * dot(n_xy_e0, vec2(v0.x, v0.y))) + max(0.0f, info.unitlength*n_xy_e0[0]) + max(0.0f, info.unitlength*n_xy_e0[1]);
		float d_xy_e1 = (-1.0f * dot(n_xy_e1, vec2(v1.x, v1.y))) + max(0.0f, info.unitlength*n_xy_e1[0]) + max(0.0f, info.unitlength*n_xy_e1[1]);
		float d_xy_e2 = (-1.0f * dot(n_xy_e2, vec2(v2.x, v2.y))) + max(0.0f, info.unitlength*n_xy_e2[0]) + max(0.0f, info.unitlength*n_xy_e2[1]);
		// YZ plane
		vec2 n_yz_e0 = vec2(-1.0f*e0.z, e0.y);
		vec2 n_yz_e1 = vec2(-1.0f*e1.z, e1.y);
		vec2 n_yz_e2 = vec2(-1.0f*e2.z, e2.y);
		if (n.x < 0.0f) {
			n_yz_e0 = -n_yz_e0;
			n_yz_e1 = -n_yz_e1;
			n_yz_e2 = -n_yz_e2;
		}
		float d_yz_e0 = (-1.0f * dot(n_yz_e0, vec2(v0.y, v0.z))) + max(0.0f, info.unitlength*n_yz_e0[0]) + max(0.0f, info.unitlength*n_yz_e0[1]);
		float d_yz_e1 = (-1.0f * dot(n_yz_e1, vec2(v1.y, v1.z))) + max(0.0f, info.unitlength*n_yz_e1[0]) + max(0.0f, info.unitlength*n_yz_e1[1]);
		float d_yz_e2 = (-1.0f * dot(n_yz_e2, vec2(v2.y, v2.z))) + max(0.0f, info.unitlength*n_yz_e2[0]) + max(0.0f, info.unitlength*n_yz_e2[1]);
		// ZX plane
		vec2 n_zx_e0 = vec2(-1.0f*e0.x, e0.z);
		vec2 n_zx_e1 = vec2(-1.0f*e1.x, e1.z);
		vec2 n_zx_e2 = vec2(-1.0f*e2.x, e2.z);
		if (n.y < 0.0f) {
			n_zx_e0 = -n_zx_e0;
			n_zx_e1 = -n_zx_e1;
			n_zx_e2 = -n_zx_e2;
		}
		float d_xz_e0 = (-1.0f * dot(n_zx_e0, vec2(v0.z, v0.x))) + max(0.0f, info.unitlength*n_zx_e0[0]) + max(0.0f, info.unitlength*n_zx_e0[1]);
		float d_xz_e1 = (-1.0f * dot(n_zx_e1, vec2(v1.z, v1.x))) + max(0.0f, info.unitlength*n_zx_e1[0]) + max(0.0f, info.unitlength*n_zx_e1[1]);
		float d_xz_e2 = (-1.0f * dot(n_zx_e2, vec2(v2.z, v2.x))) + max(0.0f, info.unitlength*n_zx_e2[0]) + max(0.0f, info.unitlength*n_zx_e2[1]);

		thread_id += blockDim.x * gridDim.x;
	}
	
}

void voxelize(voxinfo v, float* triangle_data){
	float* dev_triangle_data; // DEVICE pointer to triangle data
	bool* dev_voxelisation_table; // DEVICE pointer to voxelisation table

    //hipError_t cudaStatus = hipSuccess;

	// copy triangle data to GPU
	HANDLE_CUDA_ERROR(hipMalloc(&dev_triangle_data,v.n_triangles*9*sizeof(float)));
	HANDLE_CUDA_ERROR(hipMemcpy(dev_triangle_data, (void*) triangle_data, v.n_triangles*9*sizeof(float), hipMemcpyDefault));

	// allocate GPU memory for voxelization table
	//HANDLE_CUDA_ERROR(hipMalloc

	// if we pass triangle_data here directly, UVA takes care of memory transfer via DMA. Disabling for now.
	voxelize_triangle<<<256,256>>>(v,dev_triangle_data,0);
	hipDeviceSynchronize();
	CHECK_CUDA_ERROR();

    //return cudaStatus;

	
	
}
